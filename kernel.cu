#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdlib.h>
#include<math.h>
#include<time.h>

typedef struct
{
  float   real;
  float   img;
}complex;

texture<float,2,hipReadModeElementType>texRef1;                               //��������
texture<float,2,hipReadModeElementType>texRef2;

	


__global__ void initW(complex* W,int size_x)                                                                     //������ת����
{   
		float PI=atan((float)1)*4;
		int i = blockIdx.x*blockDim.x+threadIdx.x; 
		if(i<size_x/2)
		{
			W[i].real=cos(2*PI/size_x*i);   
			W[i].img=-1.0*sin(2*PI/size_x*i);
		}  
//		__syncthreads();
} 


__global__ void initW_array(complex* W,float* W_array_real,float* W_array_img,int size_x)                      //������ת���ӵ�����
{
	long long i = blockIdx.x*blockDim.x+threadIdx.x;
	long long j = blockIdx.y;                                                                              //����
	int l;  
	l = exp2f(j);
	//l=1<<j;
	if(i<size_x/2&&j<log((float)size_x)/log((float)2))
	{W_array_real[j*size_x/2+i] = W[size_x*(i%l)/2/l].real;
	 W_array_img[j*size_x/2+i] = W[size_x*(i%l)/2/l].img;}
//	__syncthreads();
}

 

__device__ complex ComplexMul(complex X_in,complex W_in)                     //������
{
	complex X_out;
	X_out.real = X_in.real*W_in.real-X_in.img*W_in.img;
	X_out.img = X_in.real*W_in.img+X_in.img*W_in.real;
	return X_out;
}

__device__ complex ComplexAdd(complex X1,complex X2)                           //������
{
	complex X_out;
	X_out.real = X1.real+X2.real;
	X_out.img = X1.img+X2.img;
	return X_out;
}


__device__ complex ComplexSub(complex X1,complex X2)                      //������
{
	complex X_out;
	X_out.real = X1.real-X2.real;
	X_out.img = X1.img-X2.img;
	return X_out;
}

__global__ void FFT_T(complex* DataIn,int size_x,complex* W,int Ns,int stage) 
{ 
	int k = blockIdx.x*blockDim.x+threadIdx.x;                 //�߳���block �е�λ��
   
			int p,q;
//			long long t1,t;
			complex Wn,Xp,XqWn;

		if( k<size_x/2)                                              //�������в��еĵ�������
		{
				p = k / Ns * Ns * 2 + k % Ns;
				q = p + Ns;
				Wn.real = tex2D( texRef1,k,stage );
				Wn.img = tex2D( texRef2,k,stage );
				XqWn = ComplexMul( DataIn[q],Wn);
				Xp = DataIn[p];
				DataIn[p] = ComplexAdd( Xp,XqWn);
				DataIn[q] = ComplexSub( Xp,XqWn) ;

		} //end if
} //end kernel



/*void   change(complex* h_odata)                                                                    //��λ��
  {   
  complex   temp;   
  unsigned   int   i=0,j=0,k=0;   
  unsigned int t1;
  for(i=0;i<size_x;i++)   
  {   
	  k=i;
	  j=0;   
	  t1=(log((double)size_x)/log((double)2))+0.5;                                                            //����  �������루+0.5��
//	  t1=t;
	  while(   (t1--)>0   )   
	  {   
		  j=j<<1;                                                                            //����һλ
		  j|=(k   &   1);                                                                   //ÿ��ȡ�������������һλ  ����ȡ��
		  k=k>>1;                                                                           //����һλ
	 }   
	  if(j>i)   
	  {   
		  temp=h_odata[i];   
		  h_odata[i]=h_odata[j];   
		  h_odata[j]=temp;   
	  }   
  }   
  }   

*/




__global__ void change(complex* d_idata,int size_x)                                                                    //��λ��
{
	int i = blockIdx.x*blockDim.x+threadIdx.x;
//	int n = blockIdx.y*blockDim.y+threadIdx.y;
//	int i = n * gridDim.x *blockDim.x+ m;
	complex   temp;
	unsigned int j=0,k=0;
	unsigned int   t;
if(i<size_x)
{
	  k=i;
	  j=0;   
	  t=(log((float)size_x)/log((float)2))+0.5;                                                            //����
	  while(   (t--)>0   )   
	  {   
		  j=j<<1;                                                                            //���һλ ����һλ
		  j|=(k   &   1);                                                                   //ÿ��ȡ�������������һλ  ����ȡ��
		  k=k>>1;                                                                           //�����ڶ�λ ����һλ
	  }   
	  	  if(j>i)   
	  {   
		  temp=d_idata[i];   
		  d_idata[i]=d_idata[j];   
		  d_idata[j]=temp;   
	  }   

}//end if

}//end kernal











int main()
{
	int size_x = 256*512;
//	printf("please input the size of the data:");
//	scanf("%d",&size_x);




/*
	t = log((double)size_x)/log((double)2);                        //2��������
	 
	if(t-(int)t!=0)
	{
		t = int(t) + 1;
		size_x =(int)pow(2,t);
	}
*/

	complex* d_idata;
	complex* W;

	complex* h_odata;
	complex* h_idata;

	float *W_array_real;
	float *W_array_img;

	int i=0,j=0;
	int length;
	length = size_x/1025+1;
		 float gpu_time[100]={0};
/*
	   FILE *fp1 = fopen("D:/gputime1.txt","w");
	   if(fp1==0)
	 	  exit(0);
*/
//for(size_x = 2,j = 0;size_x <= 131072;size_x = size_x * 2,j++)
//{	

	int height = (log((double)size_x)/log((double)2))+0.5;                               //cuda����߶�   ��������
//	int height;
//	height = t;
	int width = size_x/2;                                                        //cuda������
	int size = width * height * sizeof(float);

	//complex* h_idata = (complex*)malloc(size_x*sizeof(complex));                //��device�ﾭ��FFT֮�������
	//complex* h_odata = (complex*)malloc(size_x*sizeof(complex));                                         //��������ҪFFT����
	hipHostMalloc((void**)&h_odata,size_x*sizeof(complex));
	hipHostMalloc((void**)&h_idata,size_x*sizeof(complex));


	hipMalloc((void**)&W,size_x/2*sizeof(complex));
	hipMalloc((void**)&d_idata,size_x*sizeof(complex));

	hipMalloc((void**)&W_array_real,size);
	hipMalloc((void**)&W_array_img,size);

	hipChannelFormatDesc channelDesc1 = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);   //CUDA��������
	hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
	hipArray* d_Wdata_real;                                                                          //����cuda����
	hipArray* d_Wdata_img;
	hipMallocArray(&d_Wdata_real,&channelDesc1,width,height);
	hipMallocArray(&d_Wdata_img,&channelDesc2,width,height);

	

	dim3 block(length,1,1);
	dim3 thread(512,1,1);
	initW<<<block,thread>>>(W,size_x);
	hipDeviceSynchronize();

	dim3 block1(length,height,1);
	dim3 thread1(512,1,1);
	initW_array<<<block1,thread1>>>(W,W_array_real,W_array_img,size_x);
	hipDeviceSynchronize();

	hipMemcpyToArray(d_Wdata_real,0,0,W_array_real,size,hipMemcpyDeviceToDevice);                 
	hipMemcpyToArray(d_Wdata_img,0,0,W_array_img,size,hipMemcpyDeviceToDevice);
/*
	texRef1.addressMode[0]=hipAddressModeWrap;
	texRef1.addressMode[1]=hipAddressModeWrap;
	texRef1.filterMode=hipFilterModeLinear;
	texRef1.normalized=true;


	texRef2.addressMode[0]=hipAddressModeWrap;
	texRef2.addressMode[1]=hipAddressModeWrap;
	texRef2.filterMode=hipFilterModeLinear;
	texRef2.normalized=true;
*/

	hipBindTextureToArray(texRef1,d_Wdata_real,channelDesc1);                       //cuda����������
	hipBindTextureToArray(texRef2,d_Wdata_img,channelDesc2);

	
	for(i=0;i<size_x;i++)                                                         //��ҪFFT����
	{
		h_odata[i].real=i+1.0f;
		h_odata[i].img=0.0f;
	}

	clock_t start = clock();                                          //����gpu����ʱ��   ��ʼ

for(int m = 0;m < 100;m++){

	hipMemcpy(d_idata,h_odata,size_x*sizeof(complex),hipMemcpyHostToDevice);
	
	dim3 blocks3(length*2,1,1);
	dim3 threads3(512,1,1);
	change<<<blocks3,threads3>>>(d_idata,size_x);                  //��λ��
	hipDeviceSynchronize();


	dim3 blocks2(length,1,1);
	dim3 threads2(512,1,1);
	for ( int Ns = 1,stage = 0; Ns<size_x; Ns = Ns * 2,stage++)                           //Ns�Ǽ���
	{
		FFT_T<<<blocks2,threads2>>>(d_idata,size_x,W,Ns,stage);
		hipDeviceSynchronize();
	}
	hipMemcpy(h_idata,d_idata,size_x*sizeof(complex),hipMemcpyDeviceToHost);
}
	gpu_time[j] = clock() - start;                                          //����gpu����ʱ��  ����

/*	FILE *fp = fopen("D:/cuda3.txt","w");

		if(fp==0)
		  exit(0);
	  for(i = 0;i<size_x;i++)
	  {
		fprintf(fp,"%f+%f*i\n",h_idata[i].real,h_idata[i].img);
  	  }
	fclose(fp);
*/
//	fprintf(fp1," %f\n",gpu_time[j]/1000.0f);

	hipUnbindTexture(texRef1);                                               //�Ӵ�cuda���������İ�
	hipUnbindTexture(texRef2);

	//free(h_idata);                                                               //�ͷ��ڴ� �Դ�cuda����
	//free(h_odata);
	hipHostFree(h_odata);
	hipHostFree(h_idata);

	hipFree(W);
	hipFree(W_array_real);
	hipFree(W_array_img);
	hipFreeArray(d_Wdata_real);
	hipFreeArray(d_Wdata_img);
	hipFree(d_idata);

	hipDeviceReset();                                                           //�ͷ��߳�
//	hipDeviceReset();
//	hipDeviceReset();
//}
//	fclose(fp1);
	printf("%f\n",gpu_time[0]);
	printf("OK\n");
	getchar();
	return 0;
}

