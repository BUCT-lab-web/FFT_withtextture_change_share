#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdlib.h>
#include<math.h>
#include<time.h>

//#pragma comment(lib,"cutil64D.lib")
//#pragma comment(lib,"cutil64.lib")

# define N 16

typedef struct
{
  float   real;
  float   img;
}complex;

 //��������
texture<float,2,hipReadModeElementType>texRef1;
texture<float,2,hipReadModeElementType>texRef2;
texture<unsigned,2,hipReadModeElementType>texRef3;

 //������ת����
__global__ void initW(complex* W,int size_x)
{   
		float PI=atan((float)1)*4;
		int i = blockIdx.x*blockDim.x+threadIdx.x; 
		if(i<size_x/2)
		{
			W[i].real=cos(2*PI/size_x*i);   
			W[i].img=-1.0*sin(2*PI/size_x*i);
		}  
} 
 

//������ת���ӵ�����
__global__ void initW_array(complex* W,float* W_array_real,float* W_array_img,int size_x)                     
{
	long long i = blockIdx.x*blockDim.x+threadIdx.x;
	long long j = blockIdx.y;  //����
	int l;  
	l = exp2f(j);
	//l=1<<j;
	if(i<size_x/2&&j<log((float)size_x)/log((float)2))
	{W_array_real[j*size_x/2+i] = W[size_x*(i%l)/2/l].real;
	 W_array_img[j*size_x/2+i] = W[size_x*(i%l)/2/l].img;}
//	__syncthreads();
}

 
 //������
__device__ complex ComplexMul(complex X_in,complex W_in)                    
{
	complex X_out;
	X_out.real = X_in.real*W_in.real-X_in.img*W_in.img;
	X_out.img = X_in.real*W_in.img+X_in.img*W_in.real;
	return X_out;
}


//������
__device__ complex ComplexAdd(complex X1,complex X2)                           
{
	complex X_out;
	X_out.real = X1.real+X2.real;
	X_out.img = X1.img+X2.img;
	return X_out;
}


//������
__device__ complex ComplexSub(complex X1,complex X2)                      
{
	complex X_out;
	X_out.real = X1.real-X2.real;
	X_out.img = X1.img-X2.img;
	return X_out;
}


__global__ void FFT_T(complex* DataIn,int size_x,int Ns,int stage) 
{ 
	//�߳���block �е�λ��
	int k = blockIdx.x*blockDim.x+threadIdx.x;                
    int width = size_x/(2*N);
	int p,q,t;
	complex Wn,Xp,XqWn;

	    //�������в��еĵ�������
		if( k<size_x/2)                                             
		{
				p = k / Ns * Ns * 2 + k % Ns;
				q = p + Ns;

				t = (k/width)+stage;

				Wn.real = tex2D( texRef1,k%width,t );
				Wn.img = tex2D( texRef2,k%width,t );
				
				XqWn = ComplexMul( DataIn[q],Wn);
				Xp = DataIn[p];
				DataIn[p] = ComplexAdd( Xp,XqWn);
				DataIn[q] = ComplexSub( Xp,XqWn) ;
		} //end if
} //end kernel


__global__ void FFT_T1(complex* DataIn,int size_x) 
{ 
	//�߳���block �е�λ��
	int i = threadIdx.x;
	__shared__ complex sdata[1024];
	int j = blockIdx.x*blockDim.x;
	int k ;
	k = j + i;

    int width = size_x/(2*N);
	int p,q,t;
	int stage = 0;
	complex Wn,Xp,XqWn;
		
	    //�������в��еĵ�������
		if( k<size_x/2)                                             
		{
			sdata[i] = DataIn[i+j*2];
			sdata[i+512] = DataIn[i+j*2+512];
			__syncthreads();  
			
			for(int Ns = 1;Ns < 1024;Ns = Ns * 2)
			{
				p = i / Ns * Ns * 2 + i % Ns;
				q = p + Ns;

				t = (k/width)+stage;

				Wn.real = tex2D( texRef1,k%width,t );
				Wn.img = tex2D( texRef2,k%width,t );

				stage = stage + N;
				
				XqWn = ComplexMul( sdata[q],Wn);
			
				Xp = sdata[p];
				sdata[p] = ComplexAdd( Xp,XqWn);
				sdata[q] = ComplexSub( Xp,XqWn) ;
				__syncthreads();  
			}
			DataIn[p+j*2] = sdata[p];
			DataIn[q+j*2] = sdata[q];
		} //end if
} //end kernel

//��λ��
__global__ void change(unsigned *trans,int size_x)                                                                   
{
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int j=0,k=0;
	unsigned int   t;

	if(i<size_x)
	{
		k=i;
		j=0;   
		t=(log((float)size_x)/log((float)2))+0.5;   //����

		while(   (t--)>0   )   
		{   
			j=j<<1;  //���һλ ����һλ
			j|=(k   &   1);  //ÿ��ȡ�������������һλ  ����ȡ��
			k=k>>1;  //�����ڶ�λ ����һλ
		} 
		trans[i] = j;
	}//end if

}//end kernal


__global__ void change1(complex* d_idata,complex* d_idata1,int size_x)     
{
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned j;

	if(i<size_x)
	{
		j = tex2D( texRef3,i%65536,i/65536 );
		d_idata[j]=d_idata1[i];
	}
	
}


int main()
{
	int size_x = 16;
	int size_x1;

	complex* h_idata;
	complex* h_odata;

	complex* d_idata;
	complex* d_idata1;
	unsigned* trans;
	complex* W;
	float *W_array_real;
	float *W_array_img;

	int i=0;
	int length,t;

	t=(log((double)size_x)/log((double)2))+0.5;   

	float gpu_time = 0;

	length = size_x/1025+1;

	//cuda����߶�   ��������
	int height = (log((double)size_x)/log((double)2))+0.5;                               

	//cuda������
	int width = size_x/2;                                                        
	int size = width * height * sizeof(float);
                                     
	hipHostMalloc((void**)&h_odata,size_x*sizeof(complex));
	hipHostMalloc((void**)&h_idata,size_x*sizeof(complex));

	hipMalloc((void**)&W,size_x/2*sizeof(complex));
	hipMalloc((void**)&d_idata,size_x*sizeof(complex));
	hipMalloc((void**)&d_idata1,size_x*sizeof(complex));

	hipMalloc((void**)&trans,size_x*sizeof(unsigned));

	hipMalloc((void**)&W_array_real,size);
	hipMalloc((void**)&W_array_img,size);

	//CUDA��������
	hipChannelFormatDesc channelDesc1 = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);   
	hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
	hipChannelFormatDesc channelDesc3 = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindUnsigned);

	//����cuda����
	hipArray* d_Wdata_real;                                                                         
	hipArray* d_Wdata_img;
	hipArray* trans1;

	hipMallocArray(&d_Wdata_real,&channelDesc1,width/N,height*N);
	hipMallocArray(&d_Wdata_img,&channelDesc2,width/N,height*N);  
	hipMallocArray(&trans1,&channelDesc3,65536,size_x/65537+1);  

	dim3 block(length,1,1);
	dim3 thread(512,1,1);
	initW<<<block,thread>>>(W,size_x);

	dim3 block1(length,height,1);
	dim3 thread1(512,1,1);
	initW_array<<<block1,thread1>>>(W,W_array_real,W_array_img,size_x);

	dim3 blocks3(length*32,1,1);
	dim3 threads3(32,1,1);
	change<<<blocks3,threads3>>>(trans,size_x);

	hipMemcpyToArray(d_Wdata_real,0,0,W_array_real,size,hipMemcpyDeviceToDevice);                 
	hipMemcpyToArray(d_Wdata_img,0,0,W_array_img,size,hipMemcpyDeviceToDevice);
	hipMemcpyToArray(trans1,0,0,trans,size_x*sizeof(unsigned),hipMemcpyDeviceToDevice);
	
	//cuda����������
	hipBindTextureToArray(texRef1,d_Wdata_real,channelDesc1);                       
	hipBindTextureToArray(texRef2,d_Wdata_img,channelDesc2);
	hipBindTextureToArray(texRef3,trans1,channelDesc3);

	 //��ҪFFT����
	for(i=0;i<size_x;i++)                                                        
	{
		h_odata[i].real=i+1.0f;
		h_odata[i].img=0.0f;
	}

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	hipMemcpy(d_idata1,h_odata,size_x*sizeof(complex),hipMemcpyHostToDevice);

	dim3 blocks1(length*32,1,1);
	dim3 threads1(32,1,1);
	change1<<<blocks1,threads1>>>(d_idata,d_idata1,size_x);
hipEventRecord(start,0);
	//size_x1 = 1024;
	dim3 blocks4(length,1,1);
	dim3 threads4(512,1,1);
	FFT_T1<<<blocks4,threads4>>>(d_idata,size_x);


	dim3 blocks2(length,1,1);
	dim3 threads2(512,1,1);
	//Ns�Ǽ���
	for ( int Ns = 1024,stage = N*10; Ns<size_x; Ns = Ns * 2,stage+=N)                           
	{
		FFT_T<<<blocks2,threads2>>>(d_idata,size_x,Ns,stage);
		//hipDeviceSynchronize();
	}
hipEventRecord(stop,0);
hipEventSynchronize(stop);	
	hipMemcpy(h_idata,d_idata,size_x*sizeof(complex),hipMemcpyDeviceToHost);


	hipEventElapsedTime(&gpu_time,start,stop);

/*
	FILE *fp = fopen("D:/cuda3.txt","w");
	if(fp==0)
	exit(0);

	 for(i = 0;i<size_x;i++)
	 {
		fprintf(fp,"%f+%f*i\n",h_idata[i].real,h_idata[i].img);
  	 }
	fclose(fp);
*/


	hipEventDestroy(start);
	hipEventDestroy(stop);

	 //�ͷ��ڴ� �Դ�cuda����
	hipUnbindTexture(texRef1);                                               
	hipUnbindTexture(texRef2);
	hipUnbindTexture(texRef3);

	hipHostFree(h_odata);
	hipHostFree(h_idata);

	hipFree(W);
	hipFree(d_idata);
	hipFree(d_idata1);
	hipFree(trans);

	hipFree(W_array_real);
	hipFree(W_array_img);

	hipFreeArray(d_Wdata_real);
	hipFreeArray(d_Wdata_img);
	hipFreeArray(trans1);

	//�ͷ��߳�
	hipDeviceReset();                                                           

	printf("%f\n",gpu_time);
	printf("OK\n");
	getchar();

	return 0;
}

